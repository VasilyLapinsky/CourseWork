#include "HelpTools.h"

#include <exception>

void HandleCudaStatus(hipError_t cudaStatus)
{
	if (cudaStatus != hipSuccess)
	{
		throw std::exception("Bad cuda status! Error: " + cudaStatus);
	}
}

void HandleCudaRandStatus(hiprandStatus_t cudaStatus)
{
	if (cudaStatus != HIPRAND_STATUS_SUCCESS)
	{
		throw std::exception("Bad hiprand status! Error: " + cudaStatus);
	}
}