#include "Common/HelpTools.h"

#include <exception>
#include <string>

void HandleCudaStatus(hipError_t cudaStatus)
{
	if (cudaStatus != hipSuccess)
	{
		throw std::exception(("Bad cuda status! Error: " + std::to_string(static_cast<int>(cudaStatus))).c_str());
	}
}

void HandleCudaRandStatus(hiprandStatus_t cudaStatus)
{
	if (cudaStatus != HIPRAND_STATUS_SUCCESS)
	{
		throw std::exception(("Bad hiprand status! Error: " + std::to_string(static_cast<int>(cudaStatus))).c_str());
	}
}